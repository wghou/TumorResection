#include "hip/hip_runtime.h"
// Copyright 2018 BJTU.
// License: Private
// Author: Wenguo Hou
// This is a framework for neurosurgical sululation.
// 
// DeformationModelGPU is based on descent method.

#include"DeformationModelGPU.h"

#include"logger.h"
#include <iostream>
#include <fstream>

#include "Lib/CUDA_HYPER_TET_MESH.h"



DeformationModelGPU::DeformationModelGPU()
{
	m_model = new CUDA_HYPER_TET_MESH<float>();

	//Neo-Hookean
	m_model->stiffness_0 = 2000000;	//2000000
	m_model->stiffness_1 = 2000000;	//2000000

										//stVK
										//	stiffness_0 = 100000;	//2000000
										//	stiffness_1 = 6000000;	//2000000
										//	stiffness_k = 5000;	//2000

										//Mooney
										//	stiffness_0	= 2000000;	//2000000
										//   stiffness_1	= 2000000;	//2000000
										//   stiffness_2	= 2000000;	//2000000
										//	stiffness_3 = 0.5;



	m_model->control_mag = 100000;  //100000

										//Neo-Hookean
	m_model->stiffness_0 = 2000000;	//2000000
	m_model->stiffness_1 = 2000000;	//2000000

										//stVK

	m_model->model = NH_MODEL;
	m_model->stiffness_0 = 40000;	//2000000
	m_model->stiffness_1 = 200000;	//2000000
	m_model->stiffness_2 = 0;	//2000000
	m_model->stiffness_3 = 0.5;
	m_model->stiffness_p = 1000000;

	//m_model->model = NH_MODEL;
	//m_model->stiffness_0 = 200000;	//2000000
	//m_model->stiffness_1 = 2000000;	//2000000
	//m_model->stiffness_2 = 0;	//2000000
	//m_model->stiffness_3 = 0.5;
	//m_model->stiffness_p = 1000000;

	m_model->gravity = 0;

	//m_model->model = NH_MODEL;
	//m_model->stiffness_0 = 2000000;	//2000000
	//m_model->stiffness_1 = 20000000;	//2000000
	//m_model->stiffness_2 = 0;	//2000000
	//m_model->stiffness_3 = 0.5;
	//m_model->stiffness_p = 10000000;

	m_model->lower_bound = 0.15;
	m_model->upper_bound = 1000.0;



	//Mooney
	// stiffness_0	= 2000000;	//2000000
	// stiffness_1	= 2000000;	//2000000
	// stiffness_2	= 2000000;	//2000000
	// stiffness_3 = 0.5;


	m_model->control_mag = 1000000;
	m_model->profile_v[2] = 0.9997;
}


DeformationModelGPU::~DeformationModelGPU()
{
	if (m_model) delete m_model;
}


void DeformationModelGPU::Initialize(DfModel_Config & config)
{
	// read node coordinate frome file
	// trans -> rotate -> scale
	m_model->Read_Original_File(config.fileName.c_str(), config.indexDec);

	if(config.centralize) m_model->Centralize();
	else m_model->Translate(config.trans[0], config.trans[1], config.trans[2]);

	m_model->Rotate_X(config.rote[0] * 3.14159);
	m_model->Rotate_Y(config.rote[1] * 3.14159);
	m_model->Rotate_Z(config.rote[2] * 3.14159);

	m_model->Scale(config.scale);


	// �� fixedXYZ == 0 ʱ�����̶�
	for (int i = 0; i < m_model->number; i++)
	{
		if( (config.fixedAxisUp[0] != 0 && m_model->X[i * 3 + 0] > config.fixedAxisUp[0]) ||
			(config.fixedAxisBt[0] != 0 && m_model->X[i * 3 + 0] < config.fixedAxisBt[0]) ||
			(config.fixedAxisUp[1] != 0 && m_model->X[i * 3 + 1] > config.fixedAxisUp[1]) ||
			(config.fixedAxisBt[1] != 0 && m_model->X[i * 3 + 1] < config.fixedAxisBt[1]) ||
			(config.fixedAxisUp[2] != 0 && m_model->X[i * 3 + 2] > config.fixedAxisUp[2]) ||
			(config.fixedAxisBt[2] != 0 && m_model->X[i * 3 + 2] < config.fixedAxisBt[2]) )
			m_model->fixed[i] = 100000;	
	}

	Logger::getMainLogger().log(Logger::Level::Info, "vertex number: " + std::to_string(m_model->number) + "   tetrahedron number: " + std::to_string(m_model->tet_number), "");
	//printf("N: %d, %d\n", m_model->number, m_model->tet_number);

	m_model->Initialize(1.0f);

	m_model->Build_VN();
}


void DeformationModelGPU::timeStep(float time)
{
	m_model->Update(time, 97, vertexDir);
	m_model->Build_VN();
}


void DeformationModelGPU::Reset_More_Fixed(int select_v, float dir[])
{
	m_model->Reset_More_Fixed(select_v);
	vertexDir[0] = dir[0];
	vertexDir[1] = dir[1];
	vertexDir[2] = dir[2];
}


void DeformationModelGPU::SetExternalForce(float* externalForce)
{
	m_model->SetExternalForce(externalForce);
}


int DeformationModelGPU::getNodeNumber() { return m_model->number; }

float* DeformationModelGPU::getX() { return m_model->X; }

float* DeformationModelGPU::getVel() { return m_model->V; }

float* DeformationModelGPU::getVN() { return m_model->VN; }

int DeformationModelGPU::getTriNumber() { return m_model->t_number; }

uint16_t* DeformationModelGPU::getTriIndex() { return m_model->T; }

uint16_t* DeformationModelGPU::getTet() { return m_model->Tet; }

int DeformationModelGPU::getTetNumber() { return m_model->tet_number; }


void DeformationModelGPU::Scale(float s)
{
	m_model->Scale(s);
}

void DeformationModelGPU::Translate(float x, float y, float z)
{
	m_model->Translate(x, y, z);
}

void DeformationModelGPU::Rotate_X(float angle)
{
	m_model->Rotate_X(angle);
}

void DeformationModelGPU::Rotate_Y(float angle)
{
	m_model->Rotate_Y(angle);
}

void DeformationModelGPU::Rotate_Z(float angle)
{
	m_model->Rotate_Z(angle);
}