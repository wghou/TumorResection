// Copyright 2018 BJTU.
// License: Private
// Author: Wenguo Hou
// This is a framework for neurosurgical sululation.
// 
// DeformationModelGPU is based on descent method.

#include"DeformationModelGPU.h"

#include"Logger\include\Logger.h"
#include <iostream>
#include <fstream>

#include "Lib/CUDA_HYPER_TET_MESH.h"



DeformationModelGPU::DeformationModelGPU()
{
	m_model = new CUDA_HYPER_TET_MESH<float>();

	//Neo-Hookean
	m_model->stiffness_0 = 2000000;	//2000000
	m_model->stiffness_1 = 2000000;	//2000000

										//stVK
										//	stiffness_0 = 100000;	//2000000
										//	stiffness_1 = 6000000;	//2000000
										//	stiffness_k = 5000;	//2000

										//Mooney
										//	stiffness_0	= 2000000;	//2000000
										//   stiffness_1	= 2000000;	//2000000
										//   stiffness_2	= 2000000;	//2000000
										//	stiffness_3 = 0.5;



	m_model->control_mag = 100000;  //100000

										//Neo-Hookean
	m_model->stiffness_0 = 2000000;	//2000000
	m_model->stiffness_1 = 2000000;	//2000000

										//stVK

	m_model->model = NH_MODEL;
	m_model->stiffness_0 = 1000;	//2000000
	m_model->stiffness_1 = 50000;	//2000000
	m_model->stiffness_2 = 0;	//2000000
	m_model->stiffness_3 = 0.5;
	m_model->stiffness_p = 24000;


	//m_model->model = NH_MODEL;
	//m_model->stiffness_0 = 200000;	//2000000
	//m_model->stiffness_1 = 2000000;	//2000000
	//m_model->stiffness_2 = 0;	//2000000
	//m_model->stiffness_3 = 0.5;
	//m_model->stiffness_p = 1000000;

	m_model->gravity = 0.0;
	m_model->density = 100;

	//m_model->model = NH_MODEL;
	//m_model->stiffness_0 = 2000000;	//2000000
	//m_model->stiffness_1 = 20000000;	//2000000
	//m_model->stiffness_2 = 0;	//2000000
	//m_model->stiffness_3 = 0.5;
	//m_model->stiffness_p = 10000000;

	m_model->lower_bound = 0.15;
	m_model->upper_bound = 1000.0;


	//Mooney
	// stiffness_0	= 2000000;	//2000000
	// stiffness_1	= 2000000;	//2000000
	// stiffness_2	= 2000000;	//2000000
	// stiffness_3 = 0.5;


	m_model->control_mag = 50000;
	m_model->profile_v[2] = 0.9997;
}


DeformationModelGPU::~DeformationModelGPU()
{
	if (m_model) delete m_model;
}


void DeformationModelGPU::Initialize(DfModel_Config & config)
{
	m_model->number = config.numVertex;
	memcpy(m_model->X, config.mVertices, sizeof(float) * 3 * config.numVertex);
	m_model->tet_number = config.numTet;
	memcpy(m_model->Tet, config.mTets, sizeof(uint16_t) * 4 * config.numTet);

	// �� fixedXYZ == 0 ʱ�����̶�
	//config.fixedAxisBt[1] = -1;
	for (int i = 0; i < m_model->number; i++)
	{
		if( (config.fixedAxisUp[0] != 0 && m_model->X[i * 3 + 0] > config.fixedAxisUp[0]) ||
			(config.fixedAxisBt[0] != 0 && m_model->X[i * 3 + 0] < config.fixedAxisBt[0]) ||
			(config.fixedAxisUp[1] != 0 && m_model->X[i * 3 + 1] > config.fixedAxisUp[1]) ||
			(config.fixedAxisBt[1] != 0 && m_model->X[i * 3 + 1] < config.fixedAxisBt[1]) ||
			(config.fixedAxisUp[2] != 0 && m_model->X[i * 3 + 2] > config.fixedAxisUp[2]) ||
			(config.fixedAxisBt[2] != 0 && m_model->X[i * 3 + 2] < config.fixedAxisBt[2]) )
			m_model->fixed[i] = 100000;	
	}

	// �̶� fixed
	for (int i = 0; i < config.fixedVertices.size(); i++) {
		m_model->fixed[config.fixedVertices[i]] = 100000;
	}
	//std::cout << config.fixedVertices.size();

	for (int i = 0; i < m_model->tet_number; i++) {
		m_model->stiffness_0_all[i] = 1000;	//2000000
		m_model->stiffness_1_all[i] = 50000;	//2000000
		m_model->stiffness_2_all[i] = 0;	//2000000
		m_model->stiffness_3_all[i] = 0.5;
		m_model->stiffness_p_all[i] = 24000;

		m_model->dmgV[i] = 0.0f;
	}

	m_model->Initialize(1.0f);
}

void DeformationModelGPU::timeStep(float time)
{
	static float last = time;
	m_model->Update(time - last, 97, selectVertex, vertexDir);
	last = time;
}


void DeformationModelGPU::Reset_More_Fixed(int select_v, float dir[])
{
	if (select_v >= m_model->number) return;

	m_model->Reset_More_Fixed(select_v);
	selectVertex = select_v;
	vertexDir[0] = dir[0];
	vertexDir[1] = dir[1];
	vertexDir[2] = dir[2];
}


void DeformationModelGPU::SetExternalForce(float* externalForce)
{
	//m_model->SetExternalForce(externalForce);
}

void DeformationModelGPU::setGravity(float g)
{
	m_model->gravity = g;
}

void DeformationModelGPU::setMaterialParam(int start, int end, float stf_0, float stf_1, float stf_2, float stf_3, float stf_p)
{
	if (start<0 || start>end || end > m_model->tet_number) return;

	for (int i = start; i < end; i++) {
		m_model->stiffness_0_all[i] = stf_0;
		m_model->stiffness_1_all[i] = stf_1;
		m_model->stiffness_2_all[i] = stf_2;
		m_model->stiffness_3_all[i] = stf_3;
		m_model->stiffness_p_all[i] = stf_p;
	}

	hipMemcpy(m_model->dev_stiffness_0_all, m_model->stiffness_0_all, sizeof(float)*m_model->tet_number, hipMemcpyHostToDevice);
	hipMemcpy(m_model->dev_stiffness_1_all, m_model->stiffness_1_all, sizeof(float)*m_model->tet_number, hipMemcpyHostToDevice);
	hipMemcpy(m_model->dev_stiffness_2_all, m_model->stiffness_2_all, sizeof(float)*m_model->tet_number, hipMemcpyHostToDevice);
	hipMemcpy(m_model->dev_stiffness_3_all, m_model->stiffness_3_all, sizeof(float)*m_model->tet_number, hipMemcpyHostToDevice);
	hipMemcpy(m_model->dev_stiffness_p_all, m_model->stiffness_p_all, sizeof(float)*m_model->tet_number, hipMemcpyHostToDevice);
}

void DeformationModelGPU::removeTet(uint16_t t)
{
	if (t >= m_model->tet_number) return;

	m_model->rmTet[t] = 1;
	hipMemcpy(m_model->dev_rmTet, m_model->rmTet, sizeof(uint16_t)*m_model->tet_number, hipMemcpyHostToDevice);
}

void DeformationModelGPU::updateDamageV(float *dmg, int begin, int length)
{
	if (begin < 0 || length > m_model->tet_number) return;
	
	for (int i = 0; i < length; i++) {
		m_model->dmgV[i + begin] = dmg[i];
	}
}


int DeformationModelGPU::getNodeNumber() { return m_model->number; }

float* DeformationModelGPU::getX() { return m_model->X; }

float* DeformationModelGPU::getVel() { return m_model->V; }

uint16_t* DeformationModelGPU::getTet() { return m_model->Tet; }

int DeformationModelGPU::getTetNumber() { return m_model->tet_number; }