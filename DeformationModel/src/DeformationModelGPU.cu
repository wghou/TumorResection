// Copyright 2018 BJTU.
// License: Private
// Author: Wenguo Hou
// This is a framework for neurosurgical sululation.
// 
// DeformationModelGPU is based on descent method.

#include"DeformationModelGPU.h"

#include"Logger\include\Logger.h"
#include <iostream>
#include <fstream>

#include "Lib/CUDA_HYPER_TET_MESH.h"



DeformationModelGPU::DeformationModelGPU()
{
	m_model = new CUDA_HYPER_TET_MESH<float>();

	//Neo-Hookean
	m_model->stiffness_0 = 2000000;	//2000000
	m_model->stiffness_1 = 2000000;	//2000000

										//stVK
										//	stiffness_0 = 100000;	//2000000
										//	stiffness_1 = 6000000;	//2000000
										//	stiffness_k = 5000;	//2000

										//Mooney
										//	stiffness_0	= 2000000;	//2000000
										//   stiffness_1	= 2000000;	//2000000
										//   stiffness_2	= 2000000;	//2000000
										//	stiffness_3 = 0.5;



	m_model->control_mag = 100000;  //100000

										//Neo-Hookean
	m_model->stiffness_0 = 2000000;	//2000000
	m_model->stiffness_1 = 2000000;	//2000000

										//stVK

	m_model->model = NH_MODEL;
	m_model->stiffness_0 = 1000;	//2000000
	m_model->stiffness_1 = 5000;	//2000000
	m_model->stiffness_2 = 0;	//2000000
	m_model->stiffness_3 = 0.25;
	m_model->stiffness_p = 24000;

	//m_model->model = NH_MODEL;
	//m_model->stiffness_0 = 200000;	//2000000
	//m_model->stiffness_1 = 2000000;	//2000000
	//m_model->stiffness_2 = 0;	//2000000
	//m_model->stiffness_3 = 0.5;
	//m_model->stiffness_p = 1000000;

	m_model->gravity = 0.0;
	m_model->density = 1000;

	//m_model->model = NH_MODEL;
	//m_model->stiffness_0 = 2000000;	//2000000
	//m_model->stiffness_1 = 20000000;	//2000000
	//m_model->stiffness_2 = 0;	//2000000
	//m_model->stiffness_3 = 0.5;
	//m_model->stiffness_p = 10000000;

	m_model->lower_bound = 0.15;
	m_model->upper_bound = 1000.0;



	//Mooney
	// stiffness_0	= 2000000;	//2000000
	// stiffness_1	= 2000000;	//2000000
	// stiffness_2	= 2000000;	//2000000
	// stiffness_3 = 0.5;


	m_model->control_mag = 10000;
	m_model->profile_v[2] = 0.9997;
}


DeformationModelGPU::~DeformationModelGPU()
{
	if (m_model) delete m_model;
}


void DeformationModelGPU::Initialize(DfModel_Config & config)
{
	m_model->number = config.numVertex;
	memcpy(m_model->X, config.mVertices, sizeof(float) * 3 * config.numVertex);
	m_model->tet_number = config.numTet;
	memcpy(m_model->Tet, config.mTets, sizeof(uint16_t) * 4 * config.numTet);

	// �� fixedXYZ == 0 ʱ�����̶�
	for (int i = 0; i < m_model->number; i++)
	{
		if( (config.fixedAxisUp[0] != 0 && m_model->X[i * 3 + 0] > config.fixedAxisUp[0]) ||
			(config.fixedAxisBt[0] != 0 && m_model->X[i * 3 + 0] < config.fixedAxisBt[0]) ||
			(config.fixedAxisUp[1] != 0 && m_model->X[i * 3 + 1] > config.fixedAxisUp[1]) ||
			(config.fixedAxisBt[1] != 0 && m_model->X[i * 3 + 1] < config.fixedAxisBt[1]) ||
			(config.fixedAxisUp[2] != 0 && m_model->X[i * 3 + 2] > config.fixedAxisUp[2]) ||
			(config.fixedAxisBt[2] != 0 && m_model->X[i * 3 + 2] < config.fixedAxisBt[2]) )
			m_model->fixed[i] = 100000;	
	}

	// �̶� fixed
	for (int i = 0; i < config.fixedVertices.size(); i++) {
		m_model->fixed[config.fixedVertices[i]] = 100000;
	}
	//std::cout << config.fixedVertices.size();

	m_model->Initialize(1.0f);
}


void DeformationModelGPU::timeStep(float time)
{
	m_model->Update(time, 97, vertexDir);
}


void DeformationModelGPU::Reset_More_Fixed(int select_v, float dir[])
{
	if (select_v >= m_model->number) return;

	m_model->Reset_More_Fixed(select_v);
	vertexDir[0] = dir[0];
	vertexDir[1] = dir[1];
	vertexDir[2] = dir[2];
}


void DeformationModelGPU::SetExternalForce(float* externalForce)
{
	//m_model->SetExternalForce(externalForce);
}


int DeformationModelGPU::getNodeNumber() { return m_model->number; }

float* DeformationModelGPU::getX() { return m_model->X; }

float* DeformationModelGPU::getVel() { return m_model->V; }

uint16_t* DeformationModelGPU::getTet() { return m_model->Tet; }

int DeformationModelGPU::getTetNumber() { return m_model->tet_number; }